#include <hip/hip_runtime.h>
#include <openssl/sha.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_LENGTH 100
#define SHA512_DIGEST_LENGTH 64


typedef struct {
    char password[MAX_LENGTH];
    char hash[SHA512_DIGEST_LENGTH * 2 + 1]; // Simplified for demo
} PasswordHashPair;


__device__ void cuda_hash(const char *input, char *output, int length) {
    // A simple hash function
    for (int i = 0; i < length; i++) {
        output[i] = (input[i] + 1) % 256;
    }
}


__device__ void cuda_reduce(const char *hash, char *output, int length, char *chars, int chars_len) {
    // A simple reduce function
    for (int i = 0; i < length; i++) {
        int hash_val = hash[i] % chars_len;
        output[i] = chars[hash_val];
    }
    output[length] = '\0';
}

__device__ void cuda_strcpy(char *dst, const char *src) {
    while(*src != '\0') {
        *dst = *src;
        dst++;
        src++;
    }
    *dst = '\0'; // Null-terminate the destination
}

__global__ void generateRainbowTableKernel(char *chars, int chars_len, int password_len, int chain_length, int n_chains, PasswordHashPair *pairs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_chains) {
        char current_password[MAX_LENGTH];
        char current_hash[SHA512_DIGEST_LENGTH * 2 + 1];

        // Initialize with a simple pattern or use a global seed to generate
        for (int i = 0; i < password_len; i++) {
            current_password[i] = chars[(idx + i) % chars_len];
        }
        current_password[password_len] = '\0';

        // Copy initial password to the pair
        cuda_strcpy(pairs[idx].password, current_password);

        // Perform chain generation
        for (int j = 0; j < chain_length; j++) {
            cuda_hash(current_password, current_hash, password_len);
            cuda_reduce(current_hash, current_password, password_len, chars, chars_len);
        }

        cuda_strcpy(pairs[idx].hash, current_hash);
    }
}



void generateRainbowTableCUDA(char *chars, int chars_len, int password_len, int chain_length, int n_chains) {
    PasswordHashPair *dev_pairs;
    char *dev_chars;

    size_t pairs_size = n_chains * sizeof(PasswordHashPair);
    hipMalloc((void **)&dev_pairs, pairs_size);
    hipMalloc((void **)&dev_chars, chars_len);
    hipMemcpy(dev_chars, chars, chars_len, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n_chains + threadsPerBlock - 1) / threadsPerBlock;

    generateRainbowTableKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_chars, chars_len, password_len, chain_length, n_chains, dev_pairs);

    // Copy the array back to the host
    PasswordHashPair *pairs = (PasswordHashPair *)malloc(pairs_size);
    hipMemcpy(pairs, dev_pairs, pairs_size, hipMemcpyDeviceToHost);

    // Print all generated pairs
    for (int i = 0; i < n_chains; i++) {
        printf("Password: %s, Hash: %s\n", pairs[i].password, pairs[i].hash);
    }

    // Cleanup
    hipFree(dev_pairs);
    hipFree(dev_chars);
    free(pairs);
}

int main() {
    char chars[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
    int chars_len = sizeof(chars) - 1;
    int password_len = 20;
    int chain_length = 1000;
    int n_chains = 500;

    generateRainbowTableCUDA(chars, chars_len, password_len, chain_length, n_chains);

    return 0;
}